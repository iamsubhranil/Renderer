#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matrix.h"

GPU::Buffer *buffers = NULL;

hipblasHandle_t BLAShandle = NULL;

void GPU::init() {
    hipblasCreate(&BLAShandle);
}

GPU::Buffer* GPU::Buffer::alloc(size_t size) {
    GPU::Buffer **buff = &buffers;
    while (*buff) {
        GPU::Buffer *buffer = *buff;
        if (!buffer->inuse && buffer->dim == size) {
            buffer->inuse = true;
            return buffer;
        }
        buff = &(buffer->next);
    }

    GPU::Buffer *nbuff = (GPU::Buffer *)std::malloc(sizeof(GPU::Buffer));
    nbuff->dim = size;
    nbuff->inuse = true;
    hipMalloc((void**)&nbuff->values, sizeof(double) * size);
    nbuff->next = NULL;
    *buff = nbuff;
    return nbuff;
}

void GPU::Buffer::free() {
    inuse = false;
}

GPU::Buffer *getBuffer(const size_t dim, const double *values) {
    GPU::Buffer* buff = GPU::Buffer::alloc(dim);
    hipMemcpy(buff->values, values, sizeof(double) * dim,
               hipMemcpyHostToDevice);
    return buff;
}

__global__ void cudaMultiply(const int rowsA, const int colsA, const int colsB,
                             const double *a, const double *b, double *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double value = 0;
        for (int k = 0; k < colsA; ++k) {
            value += a[row * colsA + k] * b[k * colsB + col];
        }
        c[row * colsB + col] = value;
    }
}

void GPU::multiply(const int row1, const int col1, const int col2,
                   const double *v1, const double *v2, double *out,
                   bool v1OnGpu, bool v2OnGpu, bool outOnGpu) {
    GPU::Buffer *m1buffer = NULL, *m2buffer = NULL, *outbuffer = NULL;

    double *newout = out;

    if (!v1OnGpu) {
        m1buffer = getBuffer(row1 * col1, v1);
        v1 = m1buffer->values;
    }

    if (!v2OnGpu) {
        m2buffer = getBuffer(col1 * col2, v2);
        v2 = m2buffer->values;
    }

    if (!outOnGpu) {
        outbuffer = getBuffer(row1 * col2, out);
        newout = outbuffer->values;
    }

    dim3 dimBlock(16, 64);
    dim3 dimGrid((col2 + dimBlock.x - 1) / dimBlock.x,
                 (row1 + dimBlock.y - 1) / dimBlock.y);

    // cudaMultiply<<<dimGrid, dimBlock>>>(row1, col1, col2, v1,
    //                        v2, newout);

    const double alpha = 1.0f, beta = 0.0f;
    hipblasDgemm(BLAShandle, HIPBLAS_OP_N, HIPBLAS_OP_N, col1, row1, col2, &alpha, v2,
                    col1, v1, col2, &beta, newout, col1);

    if (!outOnGpu) {
        hipMemcpy(out, newout, sizeof(double) * row1 * col2,
                   hipMemcpyDeviceToHost);
    }

    if (m1buffer) m1buffer->inuse = false;
    if (m2buffer) m2buffer->inuse = false;
    if (outbuffer) outbuffer->inuse = false;
}

__global__ void cudaNormalize(double *values, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int baseidx = idx * 4;
        values[baseidx] /= values[baseidx + 3];
        values[baseidx + 1] /= values[baseidx + 3];
        values[baseidx + 2] /= values[baseidx + 3];
        values[baseidx + 3] = 1;
    }
}

__global__ void cudaCutOff(double *values, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        if (values[idx] > 1 || values[idx] < -1) {
            values[idx] = 0;
        }
    }
}

void GPU::normalizeAndCutOff(int row1, int col1, double *mat, bool onGpu) {
    GPU::Buffer *buffer = NULL;
    double *newmat = mat;

    if (!onGpu) {
        buffer = getBuffer(row1 * col1, mat);
        newmat = buffer->values;
    }

    int threadsPerBlock = 512;
    int numBlocks = (row1 * col1 + threadsPerBlock - 1) / threadsPerBlock;

    cudaNormalize<<<numBlocks, threadsPerBlock>>>(newmat, row1 * col1 / 4);
    cudaCutOff<<<numBlocks, threadsPerBlock>>>(newmat, row1 * col1);

    if (!onGpu) {
        hipMemcpy(mat, newmat, sizeof(double) * row1 * col1,
                   hipMemcpyDeviceToHost);

        buffer->inuse = false;
    }
}

void GPU::multiply_add(double *b, const double *a, double x, int size) {

    // Perform the operation b[i] += a[i] * x using cuBLAS
    hipblasDaxpy(BLAShandle, size, &x, a, 1, b, 1);
}

void *GPU::malloc(size_t size) {
    void *ret;
    hipMalloc(&ret, size);
    return ret;
}

void GPU::free(void *mem) { hipFree(mem); }

void GPU::memcpy(void *dst, void *src, size_t siz, bool reverse) {
    if (reverse) {
        hipMemcpy(dst, src, siz, hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(dst, src, siz, hipMemcpyHostToDevice);
    }
}

void *GPU::realloc(void *ptr, size_t os, size_t ns) {
    void *newPtr;
    hipMalloc(&newPtr, ns);
    size_t lt = ns < os ? ns : os;
    hipMemcpy(newPtr, ptr, lt, hipMemcpyDeviceToDevice);
    hipFree(ptr);
    return newPtr;
}
