#include "hip/hip_runtime.h"
#include "matrix.h"
#include <hipblas.h>

struct GPUBuffer {
    int dim;
    double *values;
    bool inuse;
    GPUBuffer *next;
};

GPUBuffer *buffers = NULL;

GPUBuffer *getBuffer(const int dim, const double *values) {
    GPUBuffer **buff = &buffers;
    while (*buff) {
        GPUBuffer *buffer = *buff;
        if (!buffer->inuse && buffer->dim == dim) {
            buffer->inuse = true;
            hipMemcpy(buffer->values, values, sizeof(double) * dim,
                       hipMemcpyHostToDevice);
            return buffer;
        }
        buff = &(buffer->next);
    }
    GPUBuffer *nbuff = (GPUBuffer *)malloc(sizeof(GPUBuffer));
    nbuff->dim = dim;
    nbuff->inuse = true;
    hipMalloc(&nbuff->values, sizeof(double) * dim);
    hipMemcpy(nbuff->values, values, sizeof(double) * dim,
               hipMemcpyHostToDevice);
    nbuff->next = NULL;
    *buff = nbuff;
    return nbuff;
}

__global__ void cudaMultiply(const int rowsA, const int colsA, const int colsB,
                             const double *a, const double *b, double *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double value = 0;
        for (int k = 0; k < colsA; ++k) {
            value += a[row * colsA + k] * b[k * colsB + col];
        }
        c[row * colsB + col] = value;
    }
}

void GPU::multiply(const int row1, const int col1, const int col2,
                 const double *v1, const double *v2, double *out,
                 bool v1OnGpu, bool v2OnGpu, bool outOnGpu) {
    GPUBuffer *m1buffer = NULL, *m2buffer = NULL, *outbuffer = NULL;

    double *newout = out;

    if(!v1OnGpu) {
        m1buffer = getBuffer(row1 * col1, v1);
        v1 = m1buffer->values;
    }

    if(!v2OnGpu) {
        m2buffer = getBuffer(col1 * col2, v2);
        v2 = m2buffer->values;
    }

    if(!outOnGpu) {
        outbuffer = getBuffer(row1 * col2, out);
        newout = outbuffer->values;
    }

    dim3 dimBlock(16, 64);
    dim3 dimGrid((col2 + dimBlock.x - 1) / dimBlock.x,
                 (row1 + dimBlock.y - 1) / dimBlock.y);

    // cudaMultiply<<<dimGrid, dimBlock>>>(row1, col1, col2, v1,
    //                        v2, newout);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const double alpha = 1.0f, beta = 0.0f;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, col1, row1, col2, &alpha, v2, col1, v1, col2, &beta, newout, col1);

    hipblasDestroy(handle);

    if(!outOnGpu) {
        hipMemcpy(out, newout, sizeof(double) * row1 * col2,
                   hipMemcpyDeviceToHost);
    }

    if(m1buffer)
        m1buffer->inuse = false;
    if(m2buffer)
        m2buffer->inuse = false;
    if(outbuffer)
        outbuffer->inuse = false;
}

__global__ void cudaNormalize(double *values, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int baseidx = idx * 4;
        values[baseidx] /= values[baseidx + 3];
        values[baseidx + 1] /= values[baseidx + 3];
        values[baseidx + 2] /= values[baseidx + 3];
        values[baseidx + 3] = 1;
    }
}

__global__ void cudaCutOff(double *values, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        if (values[idx] > 1 || values[idx] < -1) {
            values[idx] = 0;
        }
    }
}

void GPU::normalizeAndCutOff(int row1, int col1, double *mat, bool onGpu) {
    GPUBuffer *buffer = NULL;
    double *newmat = mat;

    if(!onGpu) {
        buffer = getBuffer(row1 * col1, mat);
        newmat = buffer->values;
    }

    int threadsPerBlock = 512;
    int numBlocks = (row1 * col1 + threadsPerBlock - 1) / threadsPerBlock;

    cudaNormalize<<<numBlocks, threadsPerBlock>>>(newmat, row1 * col1 / 4);
    cudaCutOff<<<numBlocks, threadsPerBlock>>>(newmat, row1 * col1);

    if(!onGpu) {
        hipMemcpy(mat, newmat, sizeof(double) * row1 * col1,
               hipMemcpyDeviceToHost);

        buffer->inuse = false;
    }
}

void* GPU::malloc(size_t size) {
    void *ret;
    hipMalloc(&ret, size);
    return ret;
}

void GPU::free(void *mem) {
    hipFree(mem);
}

void GPU::memcpy(void *dst, void *src, size_t siz, bool reverse) {
    if(reverse) {
        hipMemcpy(dst, src, siz,
                hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(dst, src, siz,
                hipMemcpyHostToDevice);
    }
}

void* GPU::realloc(void *ptr, size_t os, size_t ns) {
    void *newPtr;
    hipMalloc(&newPtr, ns);
    size_t lt = ns < os ? ns : os;
    hipMemcpy(newPtr, ptr, lt, hipMemcpyDeviceToDevice);
    hipFree(ptr);
    return newPtr;
}
